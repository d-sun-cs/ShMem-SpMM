#include "CusparseSpdm.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <>

#define CHECK_CUDA(func)                                                       \
{                                                                              \
    hipError_t status = (func);                                               \
    if (status != hipSuccess) {                                               \
        printf("CUDA API failed at line %d with error: %s (%d)\n",             \
               __LINE__, hipGetErrorString(status), status);                  \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

#define CHECK_CUSPARSE(func)                                                   \
{                                                                              \
    hipsparseStatus_t status = (func);                                          \
    if (status != HIPSPARSE_STATUS_SUCCESS) {                                   \
        printf("CUSPARSE API failed at line %d with error: %s (%d)\n",         \
               __LINE__, hipsparseGetErrorString(status), status);              \
        return EXIT_FAILURE;                                                   \
    }                                                                          \
}

double CusparseSpdm::runKernelSpGEMM(float* A, int nRowsA, int nColsA, float* B, int nRowsB, int nColsB, float* C) {

    int* d_csr_offsets, * d_csr_columns;
    float* d_csr_values, * d_dense;
    CHECK_CUDA(hipMalloc((void**)&d_csr_offsets,
        (nRowsA + 1) * sizeof(int)));

    hipsparseHandle_t     handle0 = nullptr;
    hipsparseDnMatDescr_t matA0;
    hipsparseSpMatDescr_t matA;
    void* dBuffer0 = NULL;
    size_t bufferSize0 = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle0));

    // Create dense matrix A0
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matA0, nRowsA, nColsA, nColsA, A,
        HIP_R_32F, HIPSPARSE_ORDER_ROW));
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE(hipsparseCreateCsr(&matA, nRowsA, nColsA, 0,
        d_csr_offsets, NULL, NULL,
        HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
        HIPSPARSE_INDEX_BASE_ZERO, HIP_R_32F));
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseDenseToSparse_bufferSize(
        handle0, matA0, matA,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
        &bufferSize0));
    CHECK_CUDA(hipMalloc(&dBuffer0, bufferSize0));

    // execute Sparse to Dense conversion
    CHECK_CUSPARSE(hipsparseDenseToSparse_analysis(handle0, matA0, matA,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
        dBuffer0));
    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE(hipsparseSpMatGetSize(matA, &num_rows_tmp, &num_cols_tmp,
        &nnz));
    // allocate CSR column indices and values
    CHECK_CUDA(hipMalloc((void**)&d_csr_columns, nnz * sizeof(int)));
    CHECK_CUDA(hipMalloc((void**)&d_csr_values, nnz * sizeof(float)));
    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE(hipsparseCsrSetPointers(matA, d_csr_offsets, d_csr_columns,
        d_csr_values));
    // execute Sparse to Dense conversion
    CHECK_CUSPARSE(hipsparseDenseToSparse_convert(handle0, matA0, matA,
        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
        dBuffer0));

    hipsparseDnMatDescr_t matB, matC;
    hipsparseHandle_t     handle = nullptr;
    float alpha = 1.0f;
    float beta = 0.0f;
    void* dBuffer = NULL;
    size_t bufferSize = 0;
    CHECK_CUSPARSE(hipsparseCreate(&handle));

    // Create dense matrix B
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matB, nRowsB, nColsB, nColsB, B,
        HIP_R_32F, HIPSPARSE_ORDER_ROW));
    // Create dense matrix C
    CHECK_CUSPARSE(hipsparseCreateDnMat(&matC, nRowsA, nColsB, nColsB, C,
        HIP_R_32F, HIPSPARSE_ORDER_ROW));
    // allocate an external buffer if needed
    CHECK_CUSPARSE(hipsparseSpMM_bufferSize(
        handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, &bufferSize));
    CHECK_CUDA(hipMalloc(&dBuffer, bufferSize));

    // execute SpMM
    CHECK_CUSPARSE(hipsparseSpMM(handle,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        HIPSPARSE_OPERATION_NON_TRANSPOSE,
        &alpha, matA, matB, &beta, matC, HIP_R_32F,
        HIPSPARSE_SPMM_ALG_DEFAULT, dBuffer));

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matA0))
        CHECK_CUSPARSE(hipsparseDestroySpMat(matA));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matB));
    CHECK_CUSPARSE(hipsparseDestroyDnMat(matC));
    CHECK_CUSPARSE(hipsparseDestroy(handle0));
    CHECK_CUSPARSE(hipsparseDestroy(handle));

    return 0.0;
}