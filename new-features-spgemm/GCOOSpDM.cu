#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>
#include <>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include "constants.h"
#include "utils.h"
#include "GCOOSpDM.h"

#define ROW_PER_GROUP 4
#define WARP_SIZE 32 // fixed size
#define BLOCK_SIZE (2 * WARP_SIZE) // fixed size

void convertToGroupCOOFormat(float* A, int nRowsA, int nColsA,
	float*& pVals, int*& pRows, int*& pCols,
	int*& pGroupIndex, int*& pNnzPerGroup, int nGroup);
__global__ void cal_group_coo_format_nnz_kernel_cm(float* A, int nRowsA, int nColsA, int* pNnzPerGroup);
__global__ void prefix_sum_kernel2(int* src, int* dst, int n);
__global__ void convert_to_groupo_coo_format_kernel_cm(
	float* A, int nRowsA, int nColsA,
	float* pVals, int* pRows, int* pCols,
	int* pGroupIndex, int* pNnzPerGroup);
__global__ void sparse_dense_groupcoo_mat_mul_kernel(float* vals_A, int* cols_A, int* rows_A,
	int* groupIndex_A, int* nnzPerGroup_A,
	int nRowsA, int nColsA,
	float* B, int nRowsB, int nColsB,
	float* C);


double GCOOSpDM::runKernelSpGEMM(float* A, int nRowsA, int nColsA, float* B, int nRowsB, int nColsB, float* C) {
	float* pVals;
	int* pRows;
	int* pCols;
	int* pGroupIndex;
	int* pNnzPerGroup;
	int nGroup = (nRowsA + ROW_PER_GROUP - 1) / ROW_PER_GROUP;

	convertToGroupCOOFormat(A, nRowsA, nColsA,
		pVals, pRows, pCols,
		pGroupIndex, pNnzPerGroup, nGroup);
	hipDeviceSynchronize();

	dim3 grid(nGroup, (nColsB + BLOCK_SIZE - 1) / BLOCK_SIZE);
	dim3 threadBlock(BLOCK_SIZE);
	sparse_dense_groupcoo_mat_mul_kernel << <grid, threadBlock >> > (
		pVals, pRows, pCols,
		pGroupIndex, pNnzPerGroup,
		nRowsA, nColsA,
		B, nRowsB, nColsB,
		C);
	hipDeviceSynchronize();

	hipFree(pVals);
	//hipFree(pCols);
	hipFree(pRows);
	hipFree(pGroupIndex);
	return 0.0;
}

void convertToGroupCOOFormat(float* A, int nRowsA, int nColsA,
	float*& pVals, int*& pRows, int*& pCols,
	int*& pGroupIndex, int*& pNnzPerGroup, int nGroup) {
	/* Print nGroup */
	if (DEBUG) {
		printf("nGroup: %d\n\n", nGroup);
	}

	/* 1. Allocate pGroupIndex and pNnzPerGroup */
	if (DEBUG) {
		checkCudaErrors(hipMallocManaged((void**)&pGroupIndex, sizeof(int) * (nGroup + 1)));
		checkCudaErrors(hipMallocManaged((void**)&pNnzPerGroup, sizeof(int) * (nGroup + 1)));
	}
	else {
		checkCudaErrors(hipMalloc((void**)&pGroupIndex, sizeof(int) * (nGroup + 1) * 2));
		// checkCudaErrors(hipMalloc((void **) &pNnzPerGroup, sizeof(int) * (nGroup+1)));
		pNnzPerGroup = pGroupIndex + (nGroup + 1);
	}
	hipMemset(pNnzPerGroup, 0, (nGroup + 1) * sizeof(int));

	/* 2. Calculate the number of non-zero elements in each group */
	dim3 gridCal(nGroup);
	dim3 tbCal(BLOCK_SIZE);
	cal_group_coo_format_nnz_kernel_cm << <gridCal, tbCal >> > (
		A, nRowsA, nColsA,
		pNnzPerGroup);
	hipDeviceSynchronize();
	if (DEBUG) {
		printf("gpu pNnzPerGroup:\n");
		print_array(pNnzPerGroup, nGroup, 1);
	}

	/* 3. Calculate pGroupIndex with pNnzPerGroup */
	prefix_sum_kernel2 << <1, 1 >> > (pNnzPerGroup, pGroupIndex, nGroup + 1);
	hipDeviceSynchronize();
	int* nnz_h = (int*)malloc(sizeof(int) * 1);
	checkCudaErrors(hipMemcpy(nnz_h, pGroupIndex + nGroup, 1 * sizeof(int), hipMemcpyDeviceToHost));
	int nnz = nnz_h[0];
	if (DEBUG) {
		printf("nnz: %d\n", nnz);
		printf("gpu pGroupIndex:\n");
		print_array(pGroupIndex, nGroup, 1);
	}

	/* 4. Allocate pVals, pRows and pCols */
	if (DEBUG) {
		checkCudaErrors(hipMallocManaged((void**)&pVals, sizeof(float) * nnz));
		checkCudaErrors(hipMallocManaged((void**)&pRows, sizeof(int) * nnz));
		checkCudaErrors(hipMallocManaged((void**)&pCols, sizeof(int) * nnz));
	}
	else {
		checkCudaErrors(hipMalloc((void**)&pVals, sizeof(float) * nnz));
		checkCudaErrors(hipMalloc((void**)&pRows, sizeof(int) * nnz * 2));
		//checkCudaErrors(hipMalloc((void **) &pCols, sizeof(int) * nnz));
		pCols = pRows + nnz;
	}

	/* 5. Calculate pVals, pRows and pCols */
	convert_to_groupo_coo_format_kernel_cm << <gridCal, tbCal >> > (A, nRowsA, nColsA,
		pVals, pRows, pCols,
		pGroupIndex, pNnzPerGroup);
	hipDeviceSynchronize();
	if (DEBUG) {
		printf("vals:\n");
		print_array(pVals, nnz, 1);
		printf("rows:\n");
		print_array(pRows, nnz, 1);
		printf("cols:\n");
		print_array(pCols, nnz, 1);
	}

	free(nnz_h);
}

__global__ void cal_group_coo_format_nnz_kernel_cm(float* A, int nRowsA, int nColsA, int* pNnzPerGroup) {
	int startIdx = blockIdx.x * ROW_PER_GROUP;
	int nnz = 0;
	for (int i = threadIdx.x; i < nColsA; i += BLOCK_SIZE) {
		for (int j = 0; j < ROW_PER_GROUP; j++) {
			int row = j + startIdx;
			if (row >= nRowsA) {
				break;
			}
			float v = A[row * nColsA + i];
			if (v != 0.0) {
				nnz++;
			}
		}
	}
	typedef hipcub::BlockReduce<int, BLOCK_SIZE> BlockReduceT;
	__shared__ typename BlockReduceT::TempStorage temp_storage;
	int aggregate = BlockReduceT(temp_storage).Sum(nnz);
	if (threadIdx.x == 0) {
		pNnzPerGroup[blockIdx.x] = aggregate;
	}
}

__global__ void prefix_sum_kernel2(int* src, int* dst, int n) {
	if (threadIdx.x == 0 && blockIdx.x == 0) {
		dst[0] = 0;
		for (int i = 1; i < n; i++) {
			dst[i] = dst[i - 1] + src[i - 1];
		}
	}
}

__global__ void convert_to_groupo_coo_format_kernel_cm(
	float* A, int nRowsA, int nColsA,
	float* pVals, int* pRows, int* pCols,
	int* pGroupIndex, int* pNnzPerGroup) {

	int startIdx = blockIdx.x * ROW_PER_GROUP;
	int currGroupOffset = pGroupIndex[blockIdx.x];
	int cooIndex = currGroupOffset;
	float* currVals = pVals + cooIndex;
	int* currCols = pCols + cooIndex;
	int* currRows = pRows + cooIndex;

	__shared__ float sA[BLOCK_SIZE * ROW_PER_GROUP];
	typedef hipcub::BlockScan<int, BLOCK_SIZE> BlockScanT;
	__shared__ typename BlockScanT::TempStorage temp_storage;

	__shared__ int sNNz;
	sNNz = 0;
	__syncthreads();

	int end = (nColsA + BLOCK_SIZE - 1) / BLOCK_SIZE * BLOCK_SIZE;
	for (int i = threadIdx.x; i < end; i += BLOCK_SIZE) {
		int nnz = 0;
		int nnz_i = 0;
		/*
		*  Calculate the number of non-zero elements in the current block position.
		*  Each thread calculate the number of nnzs in its column(ROW_PER_GROUP).
		*/
		for (int j = 0; j < ROW_PER_GROUP; j++) {
			int row = j + startIdx;
			if (row < nRowsA && i < nColsA) {
				float v = A[row * nColsA + i];
				sA[j * BLOCK_SIZE + threadIdx.x] = v;
				if (v != 0.0) {
					nnz++;
				}
			}
		}
		BlockScanT(temp_storage).InclusiveSum(nnz, nnz_i);
		__syncthreads();
		BlockScanT(temp_storage).ExclusiveSum(nnz, nnz);

		float* vals = currVals + nnz;
		int* cols = currCols + nnz;
		int* rows = currRows + nnz;
		/*
		*  Fill in GCOO arrays in column-major way,
		*  so that continuous nnz in the same column can be accessed in the following algo.
		*/
		for (int j = 0; j < ROW_PER_GROUP; j++) {
			int row = j + startIdx;
			if (row >= nRowsA || i >= nColsA)
				break;
			float v = sA[j * BLOCK_SIZE + threadIdx.x];
			if (v != 0.0) {
				*(vals++) = v;
				*(rows++) = row;
				*(cols++) = i;
			}
		}
		if (threadIdx.x == BLOCK_SIZE - 1) {
			sNNz = nnz_i;
		}
		__syncthreads();
		currVals += sNNz;
		currCols += sNNz;
		currRows += sNNz;
	}
}

__global__ void sparse_dense_groupcoo_mat_mul_kernel(float* vals_A, int* rows_A, int* cols_A,
	int* groupIndex_A, int* nnzPerGroup_A,
	int nRowsA, int nColsA,
	float* B, int nRowsB, int nColsB,
	float* C) {
	int Cj = blockIdx.y * BLOCK_SIZE + threadIdx.x;
	int Ci0 = blockIdx.x * ROW_PER_GROUP;
	float cx[ROW_PER_GROUP] = { 0.0f };
	int groupIdxOfCurrentBlock = groupIndex_A[blockIdx.x];
	int nnz = nnzPerGroup_A[blockIdx.x];
	float* currValsA = vals_A + groupIdxOfCurrentBlock;
	int* currColsA = cols_A + groupIdxOfCurrentBlock;
	int* currRowsA = rows_A + groupIdxOfCurrentBlock;

	__shared__ float sValsA[BLOCK_SIZE];
	__shared__ int sRowsA[BLOCK_SIZE];
	__shared__ int sColsA[BLOCK_SIZE];
	__shared__ int sNNz[1];

	int nIter = (BLOCK_SIZE + nnz - 1) / BLOCK_SIZE;
	int extra = nnz & (BLOCK_SIZE - 1);

	for (int i = 0; i < nIter; i++) {
		/*sColsA[threadIdx.x] = -1;
		sValsA[threadIdx.x] = 0.0;*/
		sNNz[0] = BLOCK_SIZE;
		__syncthreads();


		int valIdxStart = i * BLOCK_SIZE;
		int valIdx = valIdxStart + threadIdx.x;
		if (valIdx < nnz) {
			sValsA[threadIdx.x] = currValsA[valIdx];
			sRowsA[threadIdx.x] = currRowsA[valIdx];
			sColsA[threadIdx.x] = currColsA[valIdx];
		}
		else {
			sNNz[0] = extra;
		}
		__syncthreads();

		if (Cj < nColsB) {
			int rNNz = sNNz[0];
			int precol = -1;
			float b;
			for (int j = 0; j < rNNz; j++) {
				int col = sColsA[j];
				if (col != precol) {
					b = B[col * nColsB + Cj];
					precol = col;
				}
				float a = sValsA[j];
				int currRow = sRowsA[j];
				int index = currRow & (ROW_PER_GROUP - 1);
				// Each thread share a group of A's rows and process one column of B and C,
				// and each column contains ROW_PER_GROUP elements, represented by cx[idx]
				cx[index] += a * b;
			}
		}
		__syncthreads();
	}
	if (Cj < nColsB) {
		for (int i = 0; i < ROW_PER_GROUP && Ci0 + i < nRowsA; i++) {
			C[Cj + (Ci0 + i) * nColsB] = cx[i];
		}
	}

}